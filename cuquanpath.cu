#include "hip/hip_runtime.h"
#include "cuquanpath.h"

void QuanPath(QCircuit &qc, Matrix<DTYPE> &hostSv, int numBlocks, int numHighQubits, int numLowQubits)
{

    // 为设备分配状态向量的内存空间
    Matrix<DTYPE> *deviceSv;
    HANDLE_CUDA_ERROR(Matrix<DTYPE>::allocateDeviceMemory(deviceSv, hostSv));

    // auto start = chrono::high_resolution_clock::now();
    // Step 1. Calculate the high-order operation matrix in cpu
    Matrix<DTYPE> Opmat = highOMSim(qc, numHighQubits);

    // Step 2. Local SVSim for gates on low-order qubits
    Matrix<DTYPE> *deviceGm;
    Matrix<DTYPE> gateMatrix = Matrix<DTYPE>(4, 4);
    HANDLE_CUDA_ERROR(Matrix<DTYPE>::allocateDeviceMemory(deviceGm, gateMatrix));

    // int threadPerBlock = 256;
    // int blockPerGrid = (hostSv.row + threadPerBlock - 1) / threadPerBlock;
    int threadPerBlock = (hostSv.row + numBlocks - 1) / numBlocks;
    int blockPerGrid = numBlocks;
    threadPerBlock = threadPerBlock < 16 ? 16 : threadPerBlock;
    threadPerBlock = threadPerBlock > 1024 ? 1024 : threadPerBlock;

    for (int lev = 0; lev < qc.numDepths; ++lev)
    {
        for (int qid = 0; qid < numLowQubits; ++qid)
        {
            QGate &gate = qc.gates[lev][qid];
            if (gate.isIDE() || gate.isMARK())
            {
                continue;
            }
            gateMatrix = getCompleteMatrix(gate);
            HANDLE_CUDA_ERROR(Matrix<DTYPE>::copyHostToDevice(gateMatrix, deviceGm));
            if (gate.isSingle())
                SVSimForSingleQubit<<<blockPerGrid, threadPerBlock>>>(deviceGm, numLowQubits, deviceSv, gate.targetQubits[0]);
            else if (gate.numControls() != 0)
            {
                int q0 = gate.controlQubits[0], q1 = gate.targetQubits[0];
                SVSimForTwoQubit<<<blockPerGrid, threadPerBlock>>>(deviceGm, numLowQubits, deviceSv, min(q0, q1), max(q0, q1));
            }
            else
            {
                int q0 = gate.targetQubits[0], q1 = gate.targetQubits[1];
                SVSimForTwoQubit<<<blockPerGrid, threadPerBlock>>>(deviceGm, numLowQubits, deviceSv, min(q0, q1), max(q0, q1));
            }
        }
    }

    hipDeviceSynchronize();

    // HANDLE_CUDA_ERROR(Matrix<DTYPE>::copyDeviceToHost(deviceSv, hostSv));

    // 释放设备内存
    // HANDLE_CUDA_ERROR(Matrix<DTYPE>::freeDeviceMemory(deviceSv));
    HANDLE_CUDA_ERROR(Matrix<DTYPE>::freeDeviceMemory(deviceGm));

    // Step 3. Final merge that requires communication
    // dim3 mergegrid(1);
    // dim3 mergeblock(hostSv.row / numThreads, numThreads);

    Matrix<DTYPE> *ptrOpmat;
    // HANDLE_CUDA_ERROR(Matrix<DTYPE>::allocateDeviceMemory(deviceSv, hostSv));
    HANDLE_CUDA_ERROR(Matrix<DTYPE>::allocateDeviceMemory(ptrOpmat, Opmat));

    merge<<<blockPerGrid, 128>>>(deviceSv, ptrOpmat);
    // auto end = chrono::high_resolution_clock::now();
    hipDeviceSynchronize();

    HANDLE_CUDA_ERROR(Matrix<DTYPE>::copyDeviceToHost(deviceSv, hostSv));

    HANDLE_CUDA_ERROR(Matrix<DTYPE>::freeDeviceMemory(deviceSv));
    HANDLE_CUDA_ERROR(Matrix<DTYPE>::freeDeviceMemory(ptrOpmat));

    // chrono::duration<double> duration = end - start;
    // cout << "Svsim simulation completed in " << duration.count() << " seconds." << endl;
    hostSv.writeToTextFile("sv.txt");
}

/**
 * @brief [TODO] Conduct OMSim for high-order qubits using a thread
 *
 * @param qc a quantum circuit
 * @param numHighQubits the number of high-order qubits
 */
Matrix<DTYPE> highOMSim(QCircuit &qc, int numHighQubits)
{
    int numLowQubits = qc.numQubits - numHighQubits;
    Matrix<DTYPE> opmat, levelmat;
    opmat.identity(1 << numHighQubits);
    levelmat.identity(2);
    for (int j = 0; j < qc.numDepths; ++j)
    {
        int qid = qc.numQubits - 1;

        // get the highest gate matrix
        while (qc.gates[j][qid].isMARK() && qc.gates[j][qid].targetQubits[0] >= numLowQubits)
        {
            // Skip the pseudo placeholder MARK gates placed at control positions
            // when the target gate is applied to high-order qubits
            // If the target gate is applied to low-order qubits, MARK should be regarded as IDE
            --qid;
        }
        // [TODO] Calculate the operation matrix for gates applied to high-order qubits
        // [HINT] We have modified getCompleteMatrix to deal with MARK
        //        In this assignment, MARK is associated with an identity matrix
        // cout << "[TODO] Calculate the operation matrix for gates applied to high-order qubits" << endl;
        // MPI_Abort(MPI_COMM_WORLD, 1);
        levelmat = move(getCompleteMatrix(qc.gates[j][qid]));
        for (int i = qid - 1; i >= numLowQubits; --i)
        {
            if (qc.gates[j][i].isMARK() && qc.gates[j][i].targetQubits[0] >= numLowQubits)
            {
                continue;
            }
            Matrix<DTYPE> tmpmat = move(getCompleteMatrix(qc.gates[j][i]));
            levelmat = move(levelmat.tensorProduct(tmpmat));
        }
        opmat = move(levelmat * opmat);
        // ///////////////////////////////////////////////////////////////////////////
    }
    return opmat;
}

/**
 * @brief Conduct SVSim for gate on single qubit
 *
 * @param localSv the local state vector pointer
 * @param gateMatrix the gate matrix pointer
 * @param numLowQubits the number of low-order qubits
 * @param qidx the index of target qubit
 */
// __global__ void SVSimForSingleQubit(Matrix<DTYPE> *gateMatrix, int numLowQubits, Matrix<DTYPE> *localSv, int qidx)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int i = (idx / (1 << qidx)) * (1 << (qidx + 1));
//     int j = idx % (1 << qidx);
//     int p = i | j;
//     // 将 gateMatrix 加载到共享内存
//     __shared__ DTYPE sharedGateMatrix[2][2];
//     if (threadIdx.x < 4)
//     {
//         int row = threadIdx.x / 2;
//         int col = threadIdx.x % 2;
//         sharedGateMatrix[row][col] = gateMatrix->data[row][col];
//     }
//     __shared__ DTYPE sharedLocalSv[2048];
//     sharedLocalSv[p % 2048] = localSv->data[p][0];
//     sharedLocalSv[(p | 1 << qidx) % 2048] = localSv->data[p | 1 << qidx][0];
//     __syncthreads();

//     if (idx < localSv->row / 2)
//     {
//         // int i = (idx / (1 << qidx)) * (1 << (qidx + 1));
//         // int j = idx % (1 << qidx);
//         // int p = i | j;
//         DTYPE q0 = sharedLocalSv[p % 2048];
//         DTYPE q1 = sharedLocalSv[(p | 1 << qidx) % 2048];
//         sharedLocalSv[p % 2048] = hipCadd(hipCmul(sharedGateMatrix[0][0], q0), hipCmul(sharedGateMatrix[0][1], q1));
//         sharedLocalSv[(p | 1 << qidx) % 2048] = hipCadd(hipCmul(sharedGateMatrix[1][0], q0), hipCmul(sharedGateMatrix[1][1], q1));
//     }

//     localSv->data[p][0] = sharedLocalSv[p % 2048];
//     localSv->data[p | 1 << qidx][0] = sharedLocalSv[(p | 1 << qidx) % 2048];
// }

__global__ void SVSimForSingleQubit(Matrix<DTYPE> *gateMatrix, int numLowQubits, Matrix<DTYPE> *localSv, int qidx)
{
    // 将 gateMatrix 加载到共享内存
    __shared__ DTYPE sharedGateMatrix[2][2];
    if (threadIdx.x < 4)
    {
        int row = threadIdx.x / 2;
        int col = threadIdx.x % 2;
        sharedGateMatrix[row][col] = gateMatrix->data[row][col];
    }
    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < localSv->row / 2)
    {
        int i = (idx / (1 << qidx)) * (1 << (qidx + 1));
        int j = idx % (1 << qidx);
        int p = i | j;
        DTYPE q0 = localSv->data[p][0];
        DTYPE q1 = localSv->data[p | 1 << qidx][0];
        localSv->data[p][0] = hipCadd(hipCmul(sharedGateMatrix[0][0], q0), hipCmul(sharedGateMatrix[0][1], q1));
        localSv->data[p | 1 << qidx][0] = hipCadd(hipCmul(sharedGateMatrix[1][0], q0), hipCmul(sharedGateMatrix[1][1], q1));
    }
}

/**
 * @brief Conduct SVSim for gate on two qubits
 *
 * @param localSv the local state vector pointer
 * @param gateMatrix the gate matrix pointer
 * @param numLowQubits the number of low-order qubits
 * @param qlow low index of target qubit
 * @param qhigh high index of target qubit
 */
__global__ void SVSimForTwoQubit(Matrix<DTYPE> *gateMatrix, int numLowQubits, Matrix<DTYPE> *localSv, int qlow, int qhigh)
{
    // 将 gateMatrix 加载到共享内存
    __shared__ DTYPE sharedGateMatrix[4][4];
    if (threadIdx.x < 16)
    {
        int row = threadIdx.x / 4;
        int col = threadIdx.x % 4;
        sharedGateMatrix[row][col] = gateMatrix->data[row][col];
    }
    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < localSv->row / 4)
    {
        int i = (idx / (1 << (qhigh - 1))) * (1 << (qhigh + 1));
        int j = (((idx / (1 << (qhigh - 1))) << (qhigh - qlow - 1)) ^ (idx / (1 << qlow))) * (1 << (qlow + 1));
        int k = idx % (1 << qlow);
        int p = i | j | k;

        DTYPE q0 = localSv->data[p][0];
        DTYPE q1 = localSv->data[p | 1 << qlow][0];
        DTYPE q2 = localSv->data[p | 1 << qhigh][0];
        DTYPE q3 = localSv->data[p | 1 << qlow | 1 << qhigh][0];

        localSv->data[p][0] = hipCadd(hipCadd(hipCmul(sharedGateMatrix[0][0], q0), hipCmul(sharedGateMatrix[0][1], q1)), hipCadd(hipCmul(sharedGateMatrix[0][2], q2), hipCmul(sharedGateMatrix[0][3], q3)));
        localSv->data[p | (1 << qlow)][0] = hipCadd(hipCadd(hipCmul(sharedGateMatrix[1][0], q0), hipCmul(sharedGateMatrix[1][1], q1)), hipCadd(hipCmul(sharedGateMatrix[1][2], q2), hipCmul(sharedGateMatrix[1][3], q3)));
        localSv->data[p | (1 << qhigh)][0] = hipCadd(hipCadd(hipCmul(sharedGateMatrix[2][0], q0), hipCmul(sharedGateMatrix[2][1], q1)), hipCadd(hipCmul(sharedGateMatrix[2][2], q2), hipCmul(sharedGateMatrix[2][3], q3)));
        localSv->data[p | (1 << qlow) | (1 << qhigh)][0] = hipCadd(hipCadd(hipCmul(sharedGateMatrix[3][0], q0), hipCmul(sharedGateMatrix[3][1], q1)), hipCadd(hipCmul(sharedGateMatrix[3][2], q2), hipCmul(sharedGateMatrix[3][3], q3)));
    }
}

/**
 * @brief [TODO] Conduct the final merge operation in QuanPath
 *
 * @param sv the state vector
 * @param ptrOpmat the pointer to the high-order operation matrix
 */
__global__ void merge(Matrix<DTYPE> *sv, Matrix<DTYPE> *ptrOpmat)
{
    int opmatSize = ptrOpmat->col;
    // const int MAX_OPMAT_SIZE = 32;
    // __shared__ DTYPE sharedOpmat[MAX_OPMAT_SIZE][MAX_OPMAT_SIZE];

    // int row = threadIdx.x / opmatSize;
    // int col = threadIdx.x % opmatSize;
    // if (row < opmatSize && col < opmatSize)
    //     sharedOpmat[row][col] = ptrOpmat->data[row][col];

    // __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int localSvLen = sv->row / opmatSize;
    int totalThreadsNum = blockDim.x * gridDim.x;
    // Calculate the number of elements each thread should process
    int numElementsPerThread = (sv->row + totalThreadsNum - 1) / totalThreadsNum;

    // Calculate the starting index for the current thread
    int startIdx = idx * numElementsPerThread;

    // Loop through the elements this thread is responsible for
    for (int k = 0; k < numElementsPerThread; ++k)
    {
        int currentIdx = startIdx + k;

        if (currentIdx < sv->row)
        {
            DTYPE ans = make_hipDoubleComplex(0, 0);
            for (ll i = 0; i < opmatSize; i++)
            {
                ans = hipCadd(ans, hipCmul(ptrOpmat->data[currentIdx / localSvLen][i], sv->data[currentIdx % localSvLen + localSvLen * i][0]));
            }
            sv->data[currentIdx][0] = ans;
        }
    }
}